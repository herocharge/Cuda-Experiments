#include "hip/hip_runtime.h"
#include "pop_matrix.h"
#include <stdlib.h>

__global__ 
void mat_mul_kernel(unsigned int size, const float* op1, const float* op2, float* result){
    
}

void mat_mul(unsigned int size, const float* a, const float* b, float** c){
    const unsigned int num_bytes = sizeof(float) * size * size;

    // if mem not allocated, allocate required memory
    if(*c == NULL){
        *c = (float *)malloc(num_bytes);
    }

    // pointers to operands in gpu
    float* op1_gpu;
    float* op2_gpu;

    float* result_gpu;
    
    // allocate memory in the gpu
    hipMallocManaged(&op1_gpu, num_bytes);
    hipMallocManaged(&op2_gpu, num_bytes);
    hipMallocManaged(&result_gpu, num_bytes);

    // copy the operantds to GPU
    hipMemcpy(&op1_gpu, a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(&op2_gpu, b, num_bytes, hipMemcpyHostToDevice);

    const unsigned int nthreads = 256;
    const unsigned int nblocks = (size % 256) ? (size / 256 + 1) : (size / 256);

    // do the operation
    mat_mul_kernel<<<nblocks, nthreads>>>(size, op1_gpu, op2_gpu, result_gpu);

    hipMemcpy(*c, result_gpu, num_bytes, hipMemcpyDeviceToHost);

    // returning before freeing might be faster??
    hipFree(op1_gpu);
    hipFree(op2_gpu);

}