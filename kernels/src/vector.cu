#include "hip/hip_runtime.h"
#include "pop_vector.h"
#include <stdio.h>
#include <stdlib.h>

__global__
void vec_add_kernel(unsigned int size, const float* op1, const float* op2, float* result){
    unsigned int bidx = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tidx = threadIdx.x;
    unsigned int idx = bidx * bsize + tidx;
    result[idx] = op1[idx] + op2[idx];
}

__global__
void vec_mul_kernel(unsigned int size, const float* op1, const float* op2, float* result){
    unsigned int bidx = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tidx = threadIdx.x;
    unsigned int idx = bidx * bsize + tidx;
    result[idx] = op1[idx] * op2[idx];
}


void vec_add(const unsigned int size, const float* a, const float* b, float** c){

    const unsigned int num_bytes = sizeof(float) * size;

    // if mem not allocated, allocate required memory
    if(*c == NULL){
        *c = (float *)malloc(num_bytes);
    }

    // pointers to operands in gpu
    float* op1_gpu;
    float* op2_gpu;

    float* result_gpu;
    
    // allocate memory in the gpu
    hipMallocManaged(&op1_gpu, num_bytes);
    hipMallocManaged(&op2_gpu, num_bytes);
    hipMallocManaged(&result_gpu, num_bytes);

    // copy the operantds to GPU
    hipMemcpy(&op1_gpu, a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(&op2_gpu, b, num_bytes, hipMemcpyHostToDevice);

    const unsigned int nthreads = 256;
    const unsigned int nblocks = (size % 256) ? (size / 256 + 1) : (size / 256);

    // do the operation
    vec_add_kernel<<<nblocks, nthreads>>>(size, op1_gpu, op2_gpu, result_gpu);

    hipMemcpy(*c, result_gpu, num_bytes, hipMemcpyDeviceToHost);

    // returning before freeing might be faster??
    hipFree(op1_gpu);
    hipFree(op2_gpu);
}


void vec_mul(const unsigned int size, const float* a, const float* b, float** c){

    const unsigned int num_bytes = sizeof(float) * size;

    // if mem not allocated, allocate required memory
    if(*c == NULL){
        *c = (float *)malloc(num_bytes);
    }

    // pointers to operands in gpu
    float* op1_gpu;
    float* op2_gpu;

    float* result_gpu;
    
    // allocate memory in the gpu
    hipMallocManaged(&op1_gpu, num_bytes);
    hipMallocManaged(&op2_gpu, num_bytes);
    hipMallocManaged(&result_gpu, num_bytes);

    // copy the operantds to GPU
    hipMemcpy(&op1_gpu, a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(&op2_gpu, b, num_bytes, hipMemcpyHostToDevice);

    const unsigned int nthreads = 256;
    const unsigned int nblocks = (size % 256) ? (size / 256 + 1) : (size / 256);

    // do the operation
    vec_mul_kernel<<<nblocks, nthreads>>>(size, op1_gpu, op2_gpu, result_gpu);

    hipMemcpy(*c, result_gpu, num_bytes, hipMemcpyDeviceToHost);

    // returning before freeing might be faster??
    hipFree(op1_gpu);
    hipFree(op2_gpu);
}