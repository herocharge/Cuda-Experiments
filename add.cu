
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include "hip/hip_runtime_api.h"
// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x)
{
  printf("%p\n", x);
  // x = 0;
  for(int i = 0; i < 100; i++){
    printf("%f\n", x[i+10]);
  }
}

void kk(){
  float* x;
  int size = 10;
  hipMallocManaged(&x, size * sizeof(float));
  float b[] = {1.0f, 2.0f, 3.0f};
  // printf("%lld", sizeof(b));
  hipMemcpy(x, b, sizeof(b), hipMemcpyHostToDevice);
  printf("%p\n", b);

  add<<<1,1>>>(size, x);
  hipDeviceSynchronize();  
  hipFree(x);
}


// int main(void)
// {
//   float* x;
//   int size = 10;
//   cudaMallocManaged(&x, size * sizeof(float));
//   float b[] = {1.0f, 2.0f, 3.0f};
//   // printf("%lld", sizeof(b));
//   cudaMemcpy(x, b, sizeof(b), cudaMemcpyHostToDevice);
//   printf("%p\n", b);

//   add<<<1,1>>>(size, x);
//   cudaDeviceSynchronize();  
//   cudaFree(x);
//   return 0;
// }
